#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/* number of threads defined in a block */
//#define NUMTHREADS 64

/* size of the vectors */
#define DLEN 262144

/* debug mode prints the contents of the matrices after the calculation
 * 0 - deactivate debug mode
 * 1 - activate debug mode
 */
#define DEBUG 0

/* macro definition set up at compile time, deciding the data type
 * and precision to be used;
 */
#ifdef DOUBLE
#define DSIZE sizeof(double)
typedef double DTYPE;
#elif FLOAT
#define DSIZE sizeof(float)
typedef float DTYPE;
#endif

/* function that initializes the values in a vector given as paramenter,
 * and that has a definition and implementation dependent on the
 * definition of several macros in order to determine the data type of 
 * the vector;
 */
__host__ void init(DTYPE *vec, int N)
{
    int i, sign;
    DTYPE x, y;

    for (i = 0; i < N; ++i) {
        x = rand() % 100;
        y = (rand() + 1) % 100;
        sign = (-1) + (rand() % 2 * 2);
        vec[i] = sign * (x / y);
    }
}

/* function that prints the contents of a vector given as parameter,
 * and that has a definition and implementation dependent on the
 * definition of several macros in order to determine the data type of
 * the vector;
 */
__host__ void print_vec(DTYPE *vec, int N)
{
    int i;

    for (i = 0; i < N; i++) {
#ifdef DOUBLE
        printf("%lf ", vec[i]);
#elif FLOAT
        printf("%f ", vec[i]);
#endif
    }
    printf("\n");
}

/* GPU device function that D = A * B * scalar + C */
__global__ void func(DTYPE *A, DTYPE *B, DTYPE *C, DTYPE *D, DTYPE s, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    D[index] = A[index] * B[index] * s + C[index];
}

int main(int argc, char **argv)
{
    int N, i, NUMTHREADS;
    DTYPE *A, *B, *C, *D, scalar;
    DTYPE *dA, *dB, *dC, *dD;
    clock_t start, end;

    if (argc <= 1 || argc >= 4) {
        perror("program usage: <./benchmark.exe> <iterations> <num_threads");
        return -1;
    } else {
        N = atoi(argv[1]);
        NUMTHREADS = atoi(argv[2]);
    }

    A = (DTYPE *) malloc(DSIZE * DLEN);
    B = (DTYPE *) malloc(DSIZE * DLEN);
    C = (DTYPE *) malloc(DSIZE * DLEN);
    D = (DTYPE *) malloc(DSIZE * DLEN);

    hipMalloc((void **) &dA, DSIZE * DLEN);
    hipMalloc((void **) &dB, DSIZE * DLEN);
    hipMalloc((void **) &dC, DSIZE * DLEN);
    hipMalloc((void **) &dD, DSIZE * DLEN);

    srand(time(NULL));
    init(A, DLEN);
    init(B, DLEN);
    init(C, DLEN);
    scalar = ((-1) + (rand() % 2 * 2)) * (rand() % 10 + 1);

    if (DEBUG) {
        printf("A = \n");
        print_vec(A, DLEN);
        printf("B = \n");
        print_vec(B, DLEN);
        printf("C = \n");
        print_vec(C, DLEN);
#ifdef DOUBLE
        printf("scalar = %lf\n", scalar);
#elif FLOAT
        printf("scalar = %f\n", scalar);
#endif
    }

    hipMemcpy(dA, A, DSIZE * DLEN, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, DSIZE * DLEN, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, DSIZE * DLEN, hipMemcpyHostToDevice);
    
    start = clock();
    for (i = 0; i < N; ++i) {
        func<<<(DLEN / NUMTHREADS), NUMTHREADS>>>(dA, dB, dC, dD, scalar, DLEN);
        hipDeviceSynchronize();
    }
    end = clock();

    hipMemcpy(D, dD, DSIZE * DLEN, hipMemcpyDeviceToHost);
    
    if (DEBUG) {
        printf("D = \n");
        print_vec(D, DLEN);
    }

    printf("Execution time: %ldus\n", 
            ((long) (end - start) * 1000000) / CLOCKS_PER_SEC);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dD);

    free(A);
    free(B);
    free(C);
    free(D);

    return 0;
}
