#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/* number of threads defined in a block */
#define NUMTHREADS 64

/* debug mode prints the contents of the matrices after the calculation
 * 0 - deactivate debug mode
 * 1 - activate debug mode
 */
#define DEBUG 0

/* macro definition set up at compile time, deciding the data type
 * and precision to be used;
 */
#ifdef DOUBLE
#define DSIZE sizeof(double)
typedef double DTYPE;
#elif FLOAT
#define DSIZE sizeof(float)
typedef float DTYPE;
#endif

/* function that initializes the values in a matrix given as paramenter,
 * and that has a definition and implementation dependent on the
 * definition of several macros in order to determine the data type of 
 * the matrix;
 */
__host__ void init(DTYPE *mat, int N)
{
    int i, j, sign;
    DTYPE x, y;

    srand(time(NULL));

    for (i = 0; i < N; ++i) {
        for (j = 0; j < N; ++j) {
            x = rand();
            y = rand() + 1;
            sign = (-1) * (rand() % 2 + 1);
            mat[i * N + j] = sign * (x / y);
        }
    }
}

/* function that prints the contents of a matrix given as parameter,
 * and that has a definition and implementation dependent on the
 * definition of several macros in order to determine the data type of
 * the matrix;
 */
__host__ void print_mat(DTYPE *mat, int N)
{
    int i, j;

    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
#ifdef DOUBLE
            printf("%lf ", mat[i * N + j]);
#elif FLOAT
            printf("%f ", mat[i * N + j]);
#endif
        }
        printf("\n");
    }
}

/* GPU device function that executes multiplication */
__global__ void multiply(DTYPE *A, DTYPE *B, DTYPE *C, int N) {
    int i, row, col, index;

    index = blockIdx.x * blockDim.x + threadIdx.x;
    row = index / N;
    col = index % N;

    C[index] = 0.0;
    for (i = 0; i < N; ++i) {
        C[index] += A[row * N + i] * B[i * N + col];
    }
}

int main(int argc, char **argv)
{
    int N;
    DTYPE *A, *B, *C;
    DTYPE *dA, *dB, *dC;
    clock_t start, end;

    if (argc <= 1 || argc >= 3) {
        perror("program usage: <./benchmark.exe> <size>");
        return -1;
    } else {
        N = atoi(argv[1]);
    }

    A = (DTYPE *) malloc(DSIZE * N * N);
    B = (DTYPE *) malloc(DSIZE * N * N);
    C = (DTYPE *) malloc(DSIZE * N * N);

    hipMalloc((void **) &dA, DSIZE * N * N);
    hipMalloc((void **) &dB, DSIZE * N * N);
    hipMalloc((void **) &dC, DSIZE * N * N);

    init(A, N);
    init(B, N);

    if (DEBUG) {
        printf("A = \n");
        print_mat(A, N);
        printf("B = \n");
        print_mat(B, N);
    }

    hipMemcpy(dA, A, DSIZE * N * N, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, DSIZE * N * N, hipMemcpyHostToDevice);
    
    start = clock();
    multiply<<<(N * N / NUMTHREADS), NUMTHREADS>>>(dA, dB, dC, N);
    hipDeviceSynchronize();
    end = clock();

    hipMemcpy(C, dC, DSIZE * N * N, hipMemcpyDeviceToHost);
    
    if (DEBUG) {
        printf("C = \n");
        print_mat(C, N);
    }

    printf("Execution time: %ldus\n", 
            ((long) (end - start) * 1000000) / CLOCKS_PER_SEC);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(A);
    free(B);
    free(C);

    return 0;
}
